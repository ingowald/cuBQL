#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2023-2024 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

/*! \file check.cu builds all BVH variants we can, and runs some
    sanity and quality checks on the result */

#include "cuBQL/bvh.h"
#include "cuBQL/builder/hip/hip_runtime.h"
#include "cuBQL/builder/cpu.h"
#include "samples/common/CmdLine.h"
#include "samples/common/IO.h"
#include "testing/common/testRig.h"
#include "samples/common/Generator.h"
#include <set>
#include "cuBQL/traversal/shrinkingRadiusQuery.h"
#include <hip/hip_runtime.h>

namespace testing {

  using namespace cuBQL;
  using namespace cuBQL::samples;


  template<typename T>
  inline double costEstimate(box_t<T,2> b)
  {
    vec_t<double,2> dim = vec_t<double,2>(b.upper - b.lower);
    return dim.x + dim.y;
  }
  
  template<typename T>
  inline double costEstimate(box_t<T,3> b)
  {
    vec_t<double,3> dim = vec_t<double,3>(b.upper - b.lower);
    return dim.x*dim.y + dim.x*dim.z + dim.y*dim.z;
  }
  template<typename T>
  inline double costEstimate(box_t<T,4> b)
  {
    vec_t<double,4> dim = vec_t<double,4>(b.upper - b.lower);
    return
      dim.x*dim.y + dim.x*dim.z + dim.x*dim.w
      + dim.y*dim.z + dim.z*dim.w
      + dim.z*dim.w;
  }

    template<typename T, int D> bool isFloat3() { return false; };
    template<> bool isFloat3<float,3>() { return true; }
    

  template<typename T, int D>
  struct TreeChecker {
    using box_t   = cuBQL::box_t<T,D>;
    using vec_t   = cuBQL::vec_t<T,D>;
    using bvh_t   = cuBQL::bvh_t<T,D>;
    using node_t  = typename bvh_t::Node;

    TreeChecker(const std::vector<node_t> &nodes,
                const std::vector<int>    &primIDs,
                const std::vector<box_t>  &boxes)
      : nodes(nodes), primIDs(primIDs), boxes(boxes)
    {
      traverse(0);
      if (primIDsFound.size() != primIDs.size())
        throw std::runtime_error("some prims not found!?");
    }

    box_t traverse(int nodeID)
    {
      node_t node = nodes[nodeID];
      box_t bounds;
      if (node.admin.count) {
        for (int i=0;i<node.admin.count;i++) {
          int primID = primIDs[node.admin.offset+i];
          primIDsFound.insert(primID);
          bounds.extend(boxes[primID]);
        }
      } else {
        box_t lBounds = traverse(node.admin.offset+0);
        box_t rBounds = traverse(node.admin.offset+1);
        bounds.extend(lBounds);
        bounds.extend(rBounds);
      }
      if (node.bounds != bounds) {
        throw std::runtime_error("boxes do not match");
      }
      return bounds;
    }
    
    int numNodesFound = 0;
    std::set<int> primIDsFound;
    const std::vector<node_t> &nodes;
    const std::vector<int> &primIDs;
    const std::vector<box_t>    &boxes;
  };

  template<typename T, int D>
  __global__ void runQueries(uint64_t *d_numNodesVisited,
                             uint64_t *d_numPrimsVisited,
                             BinaryBVH<T,D> bvh,
                             const box_t<T,D> *boxes,
                             int numPoints)
  {
    using box_t   = cuBQL::box_t<T,D>;
    using vec_t   = cuBQL::vec_t<T,D>;
    using bvh_t   = cuBQL::bvh_t<T,D>;
    using node_t  = typename bvh_t::Node;
    
    int tid = threadIdx.x+blockIdx.x*blockDim.x;
    if (tid >= numPoints) return;

    vec_t queryPoint = boxes[tid].lower;
    uint64_t numNodesVisited = 0;
    uint64_t numPrimsVisited = 0;
    auto nodeDist = [&](const node_t &node) -> float
    {
      numNodesVisited++;
      return fSqrDistance_rd(queryPoint,node.bounds);
    };
    auto primCode = [&](uint32_t primID) {
      numPrimsVisited++;
      vec_t point = boxes[primID].lower;
      return fSqrDistance_rd(queryPoint,point);
    };
    shrinkingRadiusQuery::forEachPrim(primCode,nodeDist,bvh);
    atomicAdd((unsigned long long int *)d_numNodesVisited,
              (unsigned long long int)numNodesVisited);
    atomicAdd((unsigned long long int *)d_numPrimsVisited,
              (unsigned long long int)numPrimsVisited);
  }
                             
  template<typename T, int D>
  struct Checker {
    using vecND   = cuBQL::vec_t<double,D>;
    using box_t   = cuBQL::box_t<T,D>;
    using vec_t   = cuBQL::vec_t<T,D>;
    using bvh_t   = cuBQL::bvh_t<T,D>;
    using node_t  = typename bvh_t::Node;

    Checker(const std::vector<vecND> &doublePoints)
      : points(convert<T,D>(doublePoints))
    {
      //srand48(290374);
      box_t bbox;
      for (auto point : points)
        bbox.grow(point);
      float halfBoxScale = reduce_max(bbox.size()) * .5f / powf(doublePoints.size(),1./D);
       
      for (auto point : points) {
        vec_t halfBoxSize = halfBoxScale;
//        for (int i=0;i<D;i++)
          //halfBoxSize[i] *= drand48();
        boxes.push_back({point-halfBoxSize,point+halfBoxSize});
      }
      CUBQL_CUDA_CALL(Malloc((void **)&d_boxes,boxes.size()*sizeof(boxes[0])));
      CUBQL_CUDA_CALL(Memcpy((void*)d_boxes,boxes.data(),boxes.size()*sizeof(boxes[0]),
                             hipMemcpyDefault));
    }
    
    ~Checker()
    {
      CUBQL_CUDA_CALL(Free(d_boxes));
      d_boxes = 0;
    }

    double computeSAH_rec(const std::vector<node_t> &nodes,
                          const std::vector<int>    &primIDs,
                          int nodeID)
    {
      auto node = nodes[nodeID];
      double sum
        = costEstimate(node.bounds)*(1+node.admin.count);
      // double sum
      //   = node.admin.count
      //   ? costEstimate(node.bounds)*(1+node.admin.count)
      //   : 0.;
      if (node.admin.count == 0) {
        sum += computeSAH_rec(nodes,primIDs,node.admin.offset+0);
        sum += computeSAH_rec(nodes,primIDs,node.admin.offset+1);
      }
      return sum;
    }
    inline double computeSAH(const std::vector<node_t> &nodes,
                             const std::vector<int>    &primIDs)
    {
      return computeSAH_rec(nodes,primIDs,0) / costEstimate(nodes[0].bounds);
    }

    void runQuery(const std::string &description)
    {
      uint64_t *p_numNodesVisited = 0;
      uint64_t *p_numPrimsVisited = 0;
      CUBQL_CUDA_CALL(Malloc((void **)&p_numNodesVisited,sizeof(uint64_t)));
      CUBQL_CUDA_CALL(Malloc((void **)&p_numPrimsVisited,sizeof(uint64_t)));
      CUBQL_CUDA_CALL(Memset(p_numNodesVisited,0,sizeof(uint64_t)));
      CUBQL_CUDA_CALL(Memset(p_numPrimsVisited,0,sizeof(uint64_t)));
      int numPoints = (int)points.size();
      runQueries<T,D>
        <<<divRoundUp(numPoints,128),128>>>
        (p_numNodesVisited,
         p_numPrimsVisited,
         bvh,d_boxes,numPoints);
      uint64_t numNodesVisited;
      uint64_t numPrimsVisited;
      CUBQL_CUDA_CALL(Memcpy(&numNodesVisited,p_numNodesVisited,sizeof(numNodesVisited),
                             hipMemcpyDefault));
      CUBQL_CUDA_CALL(Memcpy(&numPrimsVisited,p_numPrimsVisited,sizeof(numPrimsVisited),
                             hipMemcpyDefault));
      printf("  --> num visits %20s : \tnodes %8s \tprims %8s\n",
             description.c_str(),
             prettyNumber(numNodesVisited).c_str(),
             prettyNumber(numPrimsVisited).c_str());
      // std::cout << " --> num NODES visited " << numNodesVisited << std::endl;
      // std::cout << " --> num PRIMS visited " << numPrimsVisited << std::endl;
      CUBQL_CUDA_CALL(Free(p_numNodesVisited));
      CUBQL_CUDA_CALL(Free(p_numPrimsVisited));
    }
                  
    template<
      typename runBuilderT,
      typename freeT,
      typename downloadT>
    void check(const runBuilderT &runBuilder,
               const freeT       &freeBVH,
               const downloadT   &download,
               const std::string &description,
               bool runQueryAsWell=true)
    {
      // std::cout << "# ----------------------- " << description << " ----------------------------"
      //           << std::endl;
      // std::cout << "# ...building '" << description << "'" << std::endl;
      runBuilder();

      if (runQueryAsWell) {
        runQuery(description);
      }
      
      // std::cout << "# ...downloading nodes" << std::endl;
      std::vector<typename bvh_t::Node> nodes;
      std::vector<int> primIDs;
      download(nodes,primIDs);
      // std::cout << "# ...freeing BVH" << std::endl;
      freeBVH();
      // std::cout << "# ...computing SAH cost\t\t\t" << std::flush;
      TreeChecker<T,D> check(nodes,primIDs,boxes);
      std::cout << "SAH(" << description << "): " << computeSAH(nodes,primIDs) << std::endl;
    }
    
    void checkHost()
    {
      auto freeBVH
        = [&]()
        {
          cuBQL::cpu::freeBVH(bvh);
          bvh = bvh_t{};
        };
      auto download
        = [&](std::vector<typename bvh_t::Node> &nodes,
              std::vector<int>                  &primIDs)
        {
          nodes.resize(bvh.numNodes);
          memcpy(nodes.data(),bvh.nodes,bvh.numNodes*sizeof(nodes[0]));
          primIDs.resize(bvh.numPrims);
          memcpy(primIDs.data(),bvh.primIDs,bvh.numPrims*sizeof(primIDs[0]));
        };
      check([&](){cuBQL::cpu::spatialMedian(bvh,boxes.data(),boxes.size(),BuildConfig());},
            freeBVH,
            download,
            "cpu::spatialMedian",false);
    }

    void checkDev()
    {
      auto freeBVH
        = [&]()
        {
          cuBQL::cuda::free(bvh);
          bvh = bvh_t{};
        };
      auto download
        = [&](std::vector<typename bvh_t::Node> &nodes,
              std::vector<int>                  &primIDs)
        {
          nodes.resize(bvh.numNodes);
          CUBQL_CUDA_CALL(Memcpy(nodes.data(),bvh.nodes,bvh.numNodes*sizeof(nodes[0]),
                                 hipMemcpyDefault));
          primIDs.resize(bvh.numPrims);
          CUBQL_CUDA_CALL(Memcpy(primIDs.data(),bvh.primIDs,bvh.numPrims*sizeof(primIDs[0]),
                                 hipMemcpyDefault));
        };
      check([&](){cuBQL::gpuBuilder(bvh,d_boxes,boxes.size(),BuildConfig());},
            freeBVH,
            download,
            "cuda::gpuBuilder");
      check([&](){cuBQL::cuda::radixBuilder(bvh,d_boxes,boxes.size(),BuildConfig());},
            freeBVH,
            download,
            "cuda::radixBuilder");
      check([&](){cuBQL::cuda::rebinRadixBuilder(bvh,d_boxes,boxes.size(),BuildConfig());},
            freeBVH,
            download,
            "cuda::rebinRadixBuilder");
      if (isFloat3<T,D>()) {
        check([&](){cuBQL::cuda::sahBuilder(bvh,d_boxes,boxes.size(),BuildConfig().enableSAH());},
              freeBVH,
              download,
              "cuda::sahBuilder");
      }
    }
    
    void run()
    {
      std::cout << "== " << cuBQL::vec_t<T,D>::typeName() << " ==" << std::endl;
      checkHost();
      checkDev();
    }

    std::vector<vec_t> points;
    std::vector<box_t> boxes;
    box_t             *d_boxes = 0;
    bvh_t bvh;
  };

  template<int D>
  void checkD(const std::string &generator, size_t numPoints)
  {
    std::cout << "############### numDims = " << D
              << "  generator = " << generator  << " numPoints = " << prettyNumber(numPoints)
              << " ############### " << std::endl;
    using vecND   = cuBQL::vec_t<double,D>;
    
    std::vector<vecND> points
      = PointGenerator<D>::createFromString(generator)
      ->generate(numPoints,290374);

    Checker<float,D>    (points).run();
    Checker<double,D>   (points).run();
    Checker<int,D>      (points).run();
    Checker<longlong,D> (points).run();
  }
  
  void usage(const std::string &error = "")
  {
    if (!error.empty())
      std::cout << "Error: " << error << "\n\n";
    std::cout << "Usage: ./cuBQL...cuBQL_checkBuilders [no options]" << std::endl;
    exit(error.empty()?0:1);
  }
      
} // ::testing

int main(int ac, char **av)
{
  std::vector<std::string> generatorStrings
    = {
    "uniform",
    "clustered",
    // "mixture .1 remap [ -10000000 ] [ 10000000 ] uniform remap [ 1000 1000 ] [ 10000000 1020 ] clustered",
    "mixture .1 remap [ -1000000 ] [ 1000000 ] uniform remap [ 100000 ] [ 100010 ] uniform",
  };
  // const std::string generatorString = "uniform";
  std::vector<int> numPointsToTest = { 100,10000,10000000 };
  for (auto numPoints: numPointsToTest) {
    for (auto generatorString : generatorStrings) {
      
      testing::checkD<2>(generatorString,numPoints);
      testing::checkD<3>(generatorString,numPoints);
      testing::checkD<4>(generatorString,numPoints);
    }      
  }

  return 0;
}


 
