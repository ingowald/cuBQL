#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2023-2024 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

/*! \file closestPointGPU.cu Implements a small demo-app that
  generates a set of data points, another set of query points, and
  then uses cuBQL to perform closest-point qeuries (ie, it finds,
  for each query point, the respectively closest data point */

// cuBQL itself, and the BVH type(s) it defines
#include "cuBQL/bvh.h"
#include "cuBQL/builder/hip/hip_runtime.h"
// some specialized query kernels for find-closest, on 'points' data
#include "cuBQL/queries/points/findClosest.h"
// helper class to generate various data distributions
#include "samples/common/Generator.h"

using namespace cuBQL;

enum { BVH_WIDTH = 8 };

typedef cuBQL::WideBVH<float,3,BVH_WIDTH> wide_bvh3f;

__global__
void computeBoxes(box3f *d_boxes, const vec3f *d_data, int numData)
{
  int tid = threadIdx.x+blockIdx.x*blockDim.x;
  if (tid >= numData) return;

  d_boxes[tid] = box3f().including(d_data[tid]);
}

__global__
void runQueries(wide_bvh3f bvh,
                const vec3f *d_data,
                const vec3f *d_queries,
                int numQueries)
{
  int tid = threadIdx.x+blockIdx.x*blockDim.x;
  if (tid >= numQueries) return;
  
  vec3f queryPoint = d_queries[tid];
  int closestID = cuBQL::points::findClosest(/* the cubql bvh we've built */
                                            bvh,
                                            /* data that this bvh was built over*/
                                            d_data,
                                            queryPoint);
  vec3f closestPoint = d_data[closestID];
  printf("[%i] closest point to (%f %f %f) is point #%i, at (%f %f %f)\n",
         tid,
         queryPoint.x,
         queryPoint.y,
         queryPoint.z,
         closestID,
         closestPoint.x,
         closestPoint.y,
         closestPoint.z);
}


int main(int, char **)
{
  int numDataPoints = 10000;
  int numQueryPoints = 20;
  /*! generate 10,000 uniformly distributed data points */
  std::vector<vec3f> dataPoints
    = cuBQL::samples::convert<float>
    (cuBQL::samples::UniformPointGenerator<3>()
     .generate(numDataPoints,290374));
  std::cout << "#cubql: generated " << dataPoints.size()
            << " data points" << std::endl;
  std::vector<vec3f> queryPoints
    = cuBQL::samples::convert<float>
    (cuBQL::samples::UniformPointGenerator<3>()
     .generate(numQueryPoints,/*seed*/1234567));
  std::cout << "#cubql: generated " << queryPoints.size()
            << " query points" << std::endl;

  vec3f *d_queryPoints = 0;
  vec3f *d_dataPoints = 0;
  box3f *d_primBounds = 0;
  CUBQL_CUDA_CALL(Malloc((void **)&d_queryPoints,queryPoints.size()*sizeof(vec3f)));
  CUBQL_CUDA_CALL(Memcpy(d_queryPoints,queryPoints.data(),
                         queryPoints.size()*sizeof(queryPoints[0]),
                         hipMemcpyDefault));
  CUBQL_CUDA_CALL(Malloc((void **)&d_dataPoints,dataPoints.size()*sizeof(vec3f)));
  CUBQL_CUDA_CALL(Memcpy(d_dataPoints,dataPoints.data(),
                         dataPoints.size()*sizeof(dataPoints[0]),
                         hipMemcpyDefault));
  CUBQL_CUDA_CALL(Malloc((void **)&d_primBounds,dataPoints.size()*sizeof(box3f)));
  computeBoxes<<<divRoundUp(numDataPoints,128),128>>>
    (d_primBounds,d_dataPoints,numDataPoints);

  // generate cuBQL bvh
  wide_bvh3f bvh;
  cuBQL::gpuBuilder(bvh,d_primBounds,numDataPoints,BuildConfig());
  runQueries<<<divRoundUp(numQueryPoints,128),128>>>
    (bvh,d_dataPoints,d_queryPoints,numQueryPoints);
  
  CUBQL_CUDA_SYNC_CHECK();
  return 0;
}
 
