#include "hip/hip_runtime.h"
// Copyright 2025 Ingo Wald
// SPDX-License-Identifier: Apache-2.0

#include "cuBQL/bvh.h"
#include "cuBQL/builder/hip/hip_runtime.h"
#include "cuBQL/queries/triangles/crossingCount/cc.h"
#include <fstream>
#include "../common/loadOBJ.h"

using cuBQL::Triangle;
using cuBQL::vec3i;
using cuBQL::vec3f;
using cuBQL::box3f;
using cuBQL::bvh3f;
using cuBQL::divRoundUp;


template<int axis, int sign>
inline __device__
/*! returns {signedCount,totalCount} */
int computeCrossingCount(vec3f  P,
                         vec3i *d_indices,
                         vec3f *d_vertices,
                         bvh3f  bvh,
                         bool   useTotalCount,
                         bool dbg = false)
{
  auto getTriangle = [d_vertices,d_indices](uint32_t primID)
  {
    vec3i idx = d_indices[primID];
    return Triangle{d_vertices[idx.x],d_vertices[idx.y],d_vertices[idx.z]};
  };

  cuBQL::triangles::CrossingCount cc;
  cuBQL::AxisAlignedRay<axis,sign> queryRay { P, CUBQL_INF };
  
  cc.runQuery(getTriangle,bvh,queryRay,0);

  if (useTotalCount)
    return cc.totalCount;
  else
    /* we're INside if we crossed MORE often going outwards */
    return cc.crossingCount > 0;
}

__global__ void d_computeVolume(float   *d_result,
                                vec3i    dims,
                                vec3i   *d_indices,
                                vec3f   *d_vertices,
                                box3f    worldBounds,
                                bvh3f    bvh,
                                bool     useTotalCount)
{
  int ix = threadIdx.x+blockIdx.x*blockDim.x; if (ix >= dims.x) return;
  int iy = threadIdx.y+blockIdx.y*blockDim.y; if (iy >= dims.y) return;
  int iz = threadIdx.z+blockIdx.z*blockDim.z; if (iz >= dims.z) return;

  bool dbg =  vec3i(ix,iy,iz) == vec3i(128,40,100);//dims/2;
  
  vec3f f = (vec3f(ix,iy,iz)+.5f) / vec3f(dims);
  vec3f P = (1.f-f)*worldBounds.lower + f*worldBounds.upper;

  /*! we trace 6 rays - one per principle axis - using the
      AxisAlignedRay rayquery. In theory, if the mesh is closed then
      these 6 calls should all agree; but in practice there's always
      some holes or double counting when rays going right through
      vertices or edges, so we just trace one ray in each direction
      and take a majority vote. */
  int numIn = 0;
  // numIn += computeCrossingCount<0,-1>(P,d_indices,d_vertices,bvh,useTotalCount);
  numIn += computeCrossingCount<0,+1>(P,d_indices,d_vertices,bvh,useTotalCount,dbg);
  // numIn += computeCrossingCount<1,-1>(P,d_indices,d_vertices,bvh,useTotalCount);
  // numIn += computeCrossingCount<1,+1>(P,d_indices,d_vertices,bvh,useTotalCount);
  // numIn += computeCrossingCount<2,-1>(P,d_indices,d_vertices,bvh,useTotalCount);
  // numIn += computeCrossingCount<2,+1>(P,d_indices,d_vertices,bvh,useTotalCount);

  d_result[ix+iy*dims.x+iz*dims.x*dims.y] = numIn;
}

template<typename T>
T *upload(const std::vector<T> &vec)
{
  T *d_vec = 0;
  hipMalloc((void**)&d_vec,vec.size()*sizeof(T));
  hipMemcpy(d_vec,vec.data(),vec.size()*sizeof(T),hipMemcpyDefault);
  return d_vec;
}

__global__ void fillBounds(box3f *d_bounds,
                           int numTriangles,
                           const vec3i *d_indices,
                           const vec3f *d_vertices)
{
  int tid = threadIdx.x+blockIdx.x*blockDim.x;
  if (tid >= numTriangles) return;
  vec3i idx = d_indices[tid];
  d_bounds[tid] = box3f()
    .extend(d_vertices[idx.x])
    .extend(d_vertices[idx.y])
    .extend(d_vertices[idx.z]);
}

cuBQL::bvh3f buildBVH(int numTriangles,
                      const vec3i *d_indices,
                      const vec3f *d_vertices)
{
  box3f *d_boxes;
  hipMalloc((void**)&d_boxes,numTriangles*sizeof(box3f));
  fillBounds<<<divRoundUp(numTriangles,1024),1024>>>
    (d_boxes,numTriangles,d_indices,d_vertices);

  std::cout << "building bvh" << std::endl;
  bvh3f bvh;
  ::cuBQL::gpuBuilder(bvh,d_boxes,numTriangles);
  std::cout << " ... done." << std::endl;
  hipFree(d_boxes);
  return bvh;
}
  
std::vector<float> computeVolume(const std::vector<vec3i> &indices,
                                 const std::vector<vec3f> &vertices,
                                 vec3i dims,
                                 box3f worldBounds,
                                 bool  useTotalCount)
{
  int numCells = dims.x*dims.y*dims.z;
  std::vector<float> result(numCells);
  float *d_result = 0;
  hipMalloc((void **)&d_result,numCells*sizeof(float));

  vec3f *d_vertices = upload(vertices);
  vec3i *d_indices  = upload(indices);
  
  bvh3f bvh = buildBVH(indices.size(),d_indices,d_vertices);

  vec3i bs(8);
  vec3i nb = divRoundUp(dims,bs);
  d_computeVolume<<<(dim3)nb,(dim3)bs>>>(d_result,dims,d_indices,d_vertices,
                             worldBounds,
                             bvh,useTotalCount);
  
  cuBQL::cuda::free(bvh);
  
  hipMemcpy(result.data(),d_result,numCells*sizeof(float),hipMemcpyDefault);
  hipFree(d_result);
  hipFree(d_indices);
  hipFree(d_vertices);
  return result;
}

void usage(const std::string &error)
{
  std::cerr << "Error : " << error << "\n\n";
  std::cout << "Usage: ./insideOutside inFile.obj -o outFilePrefix [-n maxRes]" << std::endl;
  exit(0);
}

int main(int ac, char **av)
{
  std::string inFileName = "";
  std::string outFileName = "";
  bool useTotalCount = false;
  int n = 256;
  for (int i=1;i<ac;i++) {
    const std::string arg = av[i];
    if (arg[0] != '-')
      inFileName = arg;
    else if (arg == "-o")
      outFileName = av[++i];
    else if (arg == "-tc") 
      useTotalCount = true;
    else if (arg == "-n") 
      n = std::stoi(av[++i]);
    else
      usage("unknown cmdline arg '"+arg+"'");
  }

  if (inFileName.empty()) usage("no input obj file name specified");
  if (outFileName.empty()) usage("no output volume file prefix specified");

  std::vector<vec3f> vertices;
  std::vector<vec3i> indices;
  std::cout << "loading obj file " << inFileName << std::endl;
  cuBQL::samples::loadOBJ(indices,vertices,inFileName);
  std::cout << "done, got " << indices.size() << " triangles" << std::endl;
  for (auto &v : vertices) v = v * 1000.f;
  box3f bb;
  for (auto v : vertices)
    bb.extend(v);
  PRINT(bb);
  vec3f size = bb.size();
  float max_size = reduce_max(size);
  vec3i dims = min(vec3i(n),vec3i(size/max_size*vec3f(n)+1.f));
  std::cout << "using volume dims of " << dims << std::endl;

  std::vector<float> result
    = computeVolume(indices,vertices,dims,bb,useTotalCount);
  std::ofstream out(outFileName
                    +"_"+std::to_string(dims.x)
                    +"x"+std::to_string(dims.y)
                    +"x"+std::to_string(dims.z)
                    +"_float.raw",
                    std::ios::binary);
  out.write((const char *)result.data(),
            dims.x*dims.y*dims.z*sizeof(float));
}
